
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <time.h>
#include <stdint.h>
#include <stdlib.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define W 800
#define H 600
#define MAX_OBJECTS 30
#define SAMPLES 100
#define MAX_BOUNCES 5
#define PIXELS (W*H)
#define THREADS 256
#define BLOCKS (ceil(PIXELS * 1.0) / THREADS)

/* Types */
typedef double sc; // scalar
typedef struct { sc x, y, z; } vec;
typedef struct { unsigned char r, g, b; } pix;

/* Vectors */
__device__ inline static sc dot(vec aa, vec bb)   { return aa.x*bb.x + aa.y*bb.y + aa.z*bb.z; }
__device__ inline static sc magsq(vec vv)         { return dot(vv, vv); }
__device__ inline static vec scale(vec vv, sc c)  { vec rv = { vv.x*c, vv.y*c, vv.z*c }; return rv; }
__device__ inline static vec normalize(vec vv)    { return scale(vv, rnorm3d(vv.x, vv.y, vv.z)); }
__device__ inline static vec add(vec aa, vec bb)  { vec rv = { aa.x+bb.x, aa.y+bb.y, aa.z+bb.z }; return rv; }
__device__ inline static vec sub(vec aa, vec bb)  { return add(aa, scale(bb, -1)); }
__device__ inline static vec hadamard_product(vec aa, vec bb) { vec rv = { aa.x*bb.x, aa.y*bb.y, aa.z*bb.z }; return rv; }

/* Ray-tracing types */
typedef vec color;              // So as to reuse dot(vv,vv) and scale
typedef struct { color albedo; sc reflectivity; sc fuzz; } material;
typedef struct { vec cp; material ma; sc r; } sphere;
typedef struct { sphere spheres[MAX_OBJECTS]; int nn; } world;
typedef struct { vec start; vec dir; } ray; // dir is normalized!

/* Random sampling */

__global__ void setup_kernel(hiprandState *state){
  int idx = threadIdx.x+blockDim.x*blockIdx.x;
  hiprand_init(1234, idx, 0, &state[idx]);
}

__host__ static sc random_double() { 
    return (rand() / (RAND_MAX + 1.0)); } // [0, 1)
__host__ static color random_color() {
    vec v = { random_double(), random_double(), random_double() };
    return v;
}

__device__ static sc d_random_double(hiprandState *d_randstate) { return hiprand_uniform_double(d_randstate); }
__device__ static vec d_random_vec(hiprandState *d_randstate) {
    vec v = { d_random_double(d_randstate), d_random_double(d_randstate), d_random_double(d_randstate) };
    return v;
}
__device__ static vec d_random_in_unit_sphere(hiprandState *d_randstate) {
    while (1) {
        vec v = d_random_vec(d_randstate);
        if (magsq(v) <= 1) return v;
    }
}
__device__ static vec d_random_unit_vector(hiprandState *d_randstate) { return normalize(d_random_in_unit_sphere(d_randstate)); }

/* Ray-tracing */

__device__ static color BLACK = {0, 0, 0};
__device__ static color WHITE = {1.0, 1.0, 1.0};
__device__ static color BLUE = {0.25, 0.49, 1.0};

__device__ static vec reflect(vec incoming, vec normal) {
    return sub(incoming, scale(normal, dot(incoming,normal)*2));
}

__device__ static int find_nearest_intersection(ray rr, sphere ss, sc *intersection) {
  vec center_rel = sub(rr.start, ss.cp);
  // Quadratic coefficients of parametric intersection equation.  a == 1.
  sc half_b = dot(center_rel, rr.dir);
  sc c = magsq(center_rel) - ss.r*ss.r;
  sc discrim = half_b*half_b - c;
  if (discrim < 0) return 0;
  sc sqdiscrim = sqrt(discrim);
  *intersection = (-half_b - sqdiscrim > 0 ? (-half_b - sqdiscrim)
                                           : (-half_b + sqdiscrim));
  return 1;
}

__device__ static color ray_color(hiprandState *randstate, const world *here, ray rr)
{
  color albedo = WHITE;

  for (int depth = 0; depth < MAX_BOUNCES; depth++) {
    const sphere *nearest_object = 0;
    sc nearest_t = 1/.0;
    sc intersection;

    for (int i = 0; i < here->nn; i++) {
      if (find_nearest_intersection(rr, here->spheres[i], &intersection)) {
        if (intersection < 0.00001 || intersection >= nearest_t) continue;
        nearest_t = intersection;
        nearest_object = &here->spheres[i];
      }
    }

    if (!nearest_object) {
        // Sky color
        sc a = 0.5 * (rr.dir.y + 1);
        return hadamard_product(albedo, add(scale(WHITE, 1.0-a), scale(BLUE, a)));
    }

    // Object color
    vec point = add(rr.start, scale(rr.dir, nearest_t));
    vec normal = normalize(sub(point, nearest_object->cp));
    vec dir = d_random_unit_vector(randstate);

    ray bounce = { point };
    if (nearest_object->ma.reflectivity == 0) { // Matte, regular scattering
      bounce.dir = add(normal, dir);
    } else { // Reflective metal scattering
      vec reflected = reflect(rr.dir, normal);
      bounce.dir = add(reflected, scale(dir, nearest_object->ma.fuzz * 0.99999));
    }
    bounce.dir = normalize(bounce.dir);
    rr = bounce;
    albedo = hadamard_product(albedo, nearest_object->ma.albedo);
  }
  return BLACK;
}

/* PPM6 */
/* PPM P6 file format; see <http://netpbm.sourceforge.net/doc/ppm.html> */

static void
output_header(int w, int h)
{ printf("P6\n%d %d\n255\n", w, h); }

__device__ static unsigned char
byte(double dd) { return dd > 1 ? 255 : dd < 0 ? 0 : dd * 255 + 0.5; }

static void
encode_color(pix p)
{ putchar(p.r); putchar(p.g); putchar(p.b); }

/* Rendering */


__device__ static ray get_ray(hiprandState *randstate, int w, int h, int x, int y) {
  // Camera is always at 0,0
  sc aspect = ((sc)w)/h; // Assume aspect >= 1
  sc viewport_height = 2.0;
  sc focal_length = 1.0; // Z distance of viewport
  sc viewport_width = viewport_height * aspect;

  sc pixel_width = (viewport_width / w);
  sc pixel_height = (viewport_height / h);
  sc left = viewport_width / -2.0;
  sc top = viewport_height / 2.0;

  sc px = left + (pixel_width * (x + d_random_double(randstate)));
  sc py = top - (pixel_height * (y + d_random_double(randstate)));

  vec pv = { px, py, focal_length };
  ray rr = { {0}, normalize(pv) };

  return rr;
}

__device__ static pix render_pixel(hiprandState *randstate, const world *here, int w, int h, int samples, int x, int y)
{
  color pixel_color = {0, 0, 0};
  for (int sample = 0; sample < samples; ++sample) {
    ray rr = get_ray(randstate, w, h, x, y);
    pixel_color = add(pixel_color, ray_color(randstate, here, rr));
  }
  pixel_color = scale(pixel_color, 1.0/samples);
  pix p = { byte(sqrt(pixel_color.x)), byte(sqrt(pixel_color.y)), byte(sqrt(pixel_color.z)) }; 
  return p;
}

__global__ void render_pixels(hiprandState *randstate, const world *here, int w, int h, int samples, pix *result)
{
  // COPY world + randstate

  int idx = threadIdx.x + blockDim.x*blockIdx.x;
  int x = idx % W;
  int y = idx / W;

  hiprandState state = randstate[threadIdx.x];

  if (idx < PIXELS) {
    result[y*W+x] = render_pixel(&state, here, w, h, samples, x, y);
  }
}

static void render(hiprandState *d_randstate,
                   world *h_here, int w, int h, int samples_per_pixel)
{
  clock_t start, stop;
  start = clock();

  // Copy the world to the GPU
  world *d_here;
  hipMalloc(&d_here, sizeof(world));
  hipMemcpy(d_here, h_here, sizeof(world), hipMemcpyHostToDevice);

  // Allocate space for the result
  pix *d_result;
  pix *h_result = (pix *)malloc(sizeof(color)*PIXELS);
  hipMalloc(&d_result, sizeof(pix)*PIXELS);

  stop = clock();
  //fprintf(stderr, "Alloc: %ldms (%0.1f fps)\n", (stop-start)/1000, 1000000.0/(stop-start));
  start = stop;

  // Calculate the pixels
  render_pixels<<<BLOCKS, THREADS>>>(d_randstate, d_here, w, h, samples_per_pixel, d_result);
  hipMemcpy(h_result, d_result, PIXELS * sizeof(pix), hipMemcpyDeviceToHost);

  stop = clock();
  fprintf(stderr, "Render: %ldms (%0.1f fps)\n", (stop-start)/1000, 1000000.0/(stop-start));
  start = stop;

  // Print PPM
  output_header(w, h);
  for (int y = 0; y < h; y++)
    for (int x = 0; x < w; x++)
      encode_color(h_result[y*W+x]);

  stop = clock();
  //fprintf(stderr, "PPM: %ldms (%0.1f fps)\n", (stop-start)/1000, 1000000.0/(stop-start));
  start = stop;
}

// Ground
sphere ground  = { .cp = {0,  -1000, 5}, .ma = { .albedo = {0.5, 0.5, 0.5} }, .r = 1000 };
// Sphere 1, reflective (fuzzier)
sphere sphere1 = { .cp = {-2, 1.0,   5}, .ma = { .albedo = {0.7, 0.7, 0.7}, .reflectivity = 1.0, .fuzz = 0.3 }, .r = 1 };
// Sphere 2, matte brown
sphere sphere2 = { .cp = {0,  1.0, 5},   .ma = { .albedo = {0.4, 0.2, 0.1} }, .r = 1 };
// Sphere 3, reflective
sphere sphere3 = { .cp = {2,  1.0, 5},    .ma = { .albedo = {0.5, 0.5, 0.5}, .reflectivity = 1.0, }, .r = 1 };
void scene(world *here) {
  sc ALT = -2.0;
  sc RAD = 0.2;

  here->spheres[here->nn++] = ground;
  here->spheres[here->nn++] = sphere1;
  here->spheres[here->nn++] = sphere2;
  here->spheres[here->nn++] = sphere3;

  for (int a=-2; a<=2; a++) {
    for (int b=3; b<=7; b++) {
      // Add a sphere
      sphere *s = &here->spheres[here->nn++];
      s->cp.x = a + 0.9*random_double();
      s->cp.y = RAD;
      s->cp.z = b + 0.9*random_double();
      s->r = RAD;
      s->ma.reflectivity = random_double() > 0.8;
      s->ma.fuzz = random_double();
      s->ma.albedo = random_color();
    }
  }

  for (int i=0; i<here->nn; i++) here->spheres[i].cp.y += ALT;
}

int main(int argc, char **argv) {
  hiprandState *d_randstate;
  hipMalloc(&d_randstate, sizeof(hiprandState)*THREADS);
  setup_kernel<<<1, 1024>>>(d_randstate);

  world here = {0};
  scene(&here);

  render(d_randstate, &here, W, H, SAMPLES);
  return 0;
}
