
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <time.h>
#include <stdint.h>
#include <stdlib.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define W 800
#define H 600
#define MAX_OBJECTS 30
#define SAMPLES 100
#define MAX_BOUNCES 50
#define PIXELS (W*H)
#define THREADS 100
#define BLOCKS (ceil(PIXELS * 1.0) / THREADS)

/* Types */
typedef double sc; // scalar
typedef struct { sc x, y, z; } vec;

/* Vectors */
__host__ __device__ inline static sc dot(vec aa, vec bb)   { return aa.x*bb.x + aa.y*bb.y + aa.z*bb.z; }
__host__ __device__ inline static sc magsq(vec vv)         { return dot(vv, vv); }
__host__ __device__ inline static vec scale(vec vv, sc c)  { vec rv = { vv.x*c, vv.y*c, vv.z*c }; return rv; }
__host__ __device__ inline static vec normalize(vec vv)    { return scale(vv, 1/sqrt(dot(vv, vv))); }
__host__ __device__ inline static vec add(vec aa, vec bb)  { vec rv = { aa.x+bb.x, aa.y+bb.y, aa.z+bb.z }; return rv; }
__host__ __device__ inline static vec sub(vec aa, vec bb)  { return add(aa, scale(bb, -1)); }
__host__ __device__ inline static vec hadamard_product(vec aa, vec bb) { vec rv = { aa.x*bb.x, aa.y*bb.y, aa.z*bb.z }; return rv; }

/* Ray-tracing types */
typedef vec color;              // So as to reuse dot(vv,vv) and scale
typedef struct { color albedo; sc reflectivity; sc fuzz; } material;
typedef struct { vec cp; material ma; sc r; } sphere;
typedef struct { sphere spheres[MAX_OBJECTS]; int nn; } world;
typedef struct { vec start; vec dir; } ray; // dir is normalized!

/* Random sampling */

__global__ void setup_kernel(hiprandState *state){
  int idx = threadIdx.x+blockDim.x*blockIdx.x;
  hiprand_init(1234, idx, 0, &state[idx]);
}

__host__ static sc random_double() { 
    return (rand() / (RAND_MAX + 1.0)); } // [0, 1)
__host__ static vec random_vec() {
    vec v = { random_double(), random_double(), random_double() };
    return v;
}
__host__ static color random_color() { return random_vec(); }

__device__ static sc d_random_double(hiprandState *d_randstate) { return hiprand_uniform_double(d_randstate); }
__device__ static vec d_random_vec(hiprandState *d_randstate) {
    vec v = { d_random_double(d_randstate), d_random_double(d_randstate), d_random_double(d_randstate) };
    return v;
}
__device__ static vec d_random_in_unit_sphere(hiprandState *d_randstate) {
    while (1) {
        vec v = d_random_vec(d_randstate);
        if (magsq(v) <= 1) return v;
    }
}
__device__ static vec d_random_unit_vector(hiprandState *d_randstate) { return normalize(d_random_in_unit_sphere(d_randstate)); }

/* Ray-tracing */

__device__ static color BLACK = {0, 0, 0};
__device__ static color WHITE = {1.0, 1.0, 1.0};
__device__ static color BLUE = {0.25, 0.49, 1.0};

__device__ static vec reflect(vec incoming, vec normal) {
    return sub(incoming, scale(normal, dot(incoming,normal)*2));
}

__device__ static int find_nearest_intersection(ray rr, sphere ss, sc *intersection) {
  vec center_rel = sub(rr.start, ss.cp);
  // Quadratic coefficients of parametric intersection equation.  a == 1.
  sc b = 2*dot(center_rel, rr.dir), c = magsq(center_rel) - ss.r*ss.r;
  sc discrim = b*b - 4*c;
  if (discrim < 0) return 0;
  sc sqdiscrim = sqrt(discrim);
  *intersection = (-b - sqdiscrim > 0 ? (-b - sqdiscrim)/2
                                      : (-b + sqdiscrim)/2);
  return 1;
}

__device__ static color ray_color(hiprandState *randstate, const world *here, ray rr)
{
  sc intersection;
  sc nearest_t;
  const sphere *nearest_object;
  color albedo = WHITE;

  for (int depth = 0; depth < MAX_BOUNCES; depth++) {
    nearest_object = 0;
    nearest_t = 1/.0;
    for (int i = 0; i < here->nn; i++) {
        if (find_nearest_intersection(rr, here->spheres[i], &intersection)) {
        if (intersection < 0.000001 || intersection >= nearest_t) continue;
        nearest_t = intersection;
        nearest_object = &here->spheres[i];
        }
    }

    if (nearest_object) {
        // Object color
        vec point = add(rr.start, scale(rr.dir, nearest_t));
        vec normal = normalize(sub(point, nearest_object->cp));

        ray bounce = { point };
        if (nearest_object->ma.reflectivity == 0) { // Matte, regular scattering
            bounce.dir = add(normal, d_random_unit_vector(randstate));
        } else { // Reflective metal scattering
            vec reflected = reflect(rr.dir, normal);
            bounce.dir = add(reflected, scale(d_random_unit_vector(randstate), nearest_object->ma.fuzz));
            if (dot(bounce.dir, normal) < 0) return BLACK;
        }
        if (magsq(bounce.dir) < 0.0000001) return BLACK;
        bounce.dir = normalize(bounce.dir);
        rr = bounce;
        albedo = hadamard_product(albedo, nearest_object->ma.albedo);
    } else {
        // Sky color
        sc a = 0.5 * (rr.dir.y + 1);
        return hadamard_product(albedo, add(scale(WHITE, 1.0-a), scale(BLUE, a)));
    }
  }
  return BLACK;
}

/* PPM6 */
/* PPM P6 file format; see <http://netpbm.sourceforge.net/doc/ppm.html> */

static void
output_header(int w, int h)
{ printf("P6\n%d %d\n255\n", w, h); }

static unsigned char
byte(double dd) { return dd > 1 ? 255 : dd < 0 ? 0 : dd * 255 + 0.5; }

static void
encode_color(color co)
{ putchar(byte(sqrt(co.x))); putchar(byte(sqrt(co.y))); putchar(byte(sqrt(co.z))); }

/* Rendering */


__device__ static ray get_ray(hiprandState *randstate, int w, int h, int x, int y) {
  // Camera is always at 0,0
  sc aspect = ((sc)w)/h; // Assume aspect >= 1
  sc viewport_height = 2.0;
  sc focal_length = 1.0; // Z distance of viewport
  sc viewport_width = viewport_height * aspect;

  sc pixel_width = (viewport_width / w);
  sc pixel_height = (viewport_height / h);
  sc left = viewport_width / -2.0;
  sc top = viewport_height / 2.0;

  sc px = left + (pixel_width * (x + d_random_double(randstate)));
  sc py = top - (pixel_height * (y + d_random_double(randstate)));

  vec pv = { px, py, focal_length };
  ray rr = { {0}, normalize(pv) };

  return rr;
}

__device__ static void render_pixel(hiprandState *randstate, const world *here, int w, int h, int samples, int x, int y, color *result)
{
  color pixel_color = {0, 0, 0};
  for (int sample = 0; sample < samples; ++sample) {
    ray rr = get_ray(randstate, w, h, x, y);
    pixel_color = add(pixel_color, ray_color(randstate, here, rr));
  }
  *result = pixel_color;
}

__global__ void render_pixels(hiprandState *randstate, const world *here, int w, int h, int samples, color *result)
{
  int idx = threadIdx.x + blockDim.x*blockIdx.x;
  if (idx >= PIXELS) return;
  int x = idx % W;
  int y = idx / W;

  render_pixel(randstate, here, w, h, samples, x, y, &result[y*W+x]);
}

static void render(hiprandState *d_randstate,
                   world *h_here, int w, int h, int samples_per_pixel)
{
  // Copy the world to the GPU
  world *d_here;
  hipMalloc(&d_here, sizeof(world));
  hipMemcpy(d_here, h_here, sizeof(world), hipMemcpyHostToDevice);

  // Allocate space for the result
  color *d_result;
  color *h_result = (color *)malloc(sizeof(color)*PIXELS);
  hipMalloc(&d_result, sizeof(color)*PIXELS);

  // Calculate the pixels
  render_pixels<<<BLOCKS, THREADS>>>(d_randstate, d_here, w, h, samples_per_pixel, d_result);
  hipMemcpy(h_result, d_result, PIXELS * sizeof(color), hipMemcpyDeviceToHost);

  // Print PPM
  output_header(w, h);
  for (int y = 0; y < h; y++) {
    for (int x = 0; x < w; x++) {
      color pixel_color = h_result[y*W+x];
      encode_color(scale(pixel_color, 1.0/samples_per_pixel));
    }
  }
}

// Ground
sphere ground  = { .cp = {0,  -1000, 5}, .ma = { .albedo = {0.5, 0.5, 0.5} }, .r = 1000 };
// Sphere 1, reflective (fuzzier)
sphere sphere1 = { .cp = {-2, 1.0,   5}, .ma = { .albedo = {0.7, 0.7, 0.7}, .reflectivity = 1.0, .fuzz = 0.3 }, .r = 1 };
// Sphere 2, matte brown
sphere sphere2 = { .cp = {0,  1.0, 5},   .ma = { .albedo = {0.4, 0.2, 0.1} }, .r = 1 };
// Sphere 3, reflective
sphere sphere3 = { .cp = {2,  1.0, 5},    .ma = { .albedo = {0.5, 0.5, 0.5}, .reflectivity = 1.0, }, .r = 1 };
void scene(world *here) {
  sc ALT = -2.0;
  sc RAD = 0.2;

  here->spheres[here->nn++] = ground;
  here->spheres[here->nn++] = sphere1;
  here->spheres[here->nn++] = sphere2;
  here->spheres[here->nn++] = sphere3;

  for (int a=-2; a<=2; a++) {
    for (int b=3; b<=7; b++) {
      // Add a sphere
      sphere *s = &here->spheres[here->nn++];
      s->cp.x = a + 0.9*random_double();
      s->cp.y = RAD;
      s->cp.z = b + 0.9*random_double();
      s->r = RAD;
      s->ma.albedo = random_color();
      s->ma.reflectivity = random_double() > 0.8;
      s->ma.fuzz = random_double();
    }
  }

  for (int i=0; i<here->nn; i++) here->spheres[i].cp.y += ALT;
}

int main(int argc, char **argv) {
  hiprandState *d_randstate;
  hipMalloc(&d_randstate, sizeof(hiprandState));
  setup_kernel<<<1,W>>>(d_randstate);

  world here = {0};
  scene(&here);

  clock_t start, stop;
  start = clock();
  render(d_randstate, &here, W, H, SAMPLES);
  stop = clock();
  fprintf(stderr, "Render: %ldms (%0.1f fps)\n", (stop-start)/1000, 1000000.0/(stop-start));
  return 0;
}
